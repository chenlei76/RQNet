#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "activation.h"



__global__ static void activate_kernel(float* data, int elements, int threads, ACTIVATION_TYPE a) {

	float val;
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	while (index < elements) {

		switch (a) {
		case LEAKY:
			if (data[index] < 0.0)  data[index] *= 0.1;
			break;
		case LOGISTIC:
			//__device__ float logistic_activate_kernel(float x){return 1./(1. + exp(-x));}  
			data[index] = 1.0 / (1.0 + exp(-data[index]));
			break;
		case RELU:
			if (data[index] < 0) data[index] = 0.0;
			break;
		case HARDTAN:
			if (data[index] < -1.0)
				data[index] = -1.0;
			else if (data[index] > 1.0)
				data[index] = 1.0;
			break;
		case LHTAN:
			if (data[index] < -0.0)
				data[index] *= 0.001;
			else if (data[index] > 1.0)
				data[index] = 0.001 * (data[index] - 1) + 1;
			break;
		case TANH:
			val = exp(2.0 * data[index]);
			data[index] = (val - 1) / (val + 1);
			break;
		case LOGGY:
			data[index] = 2.0 / (1.0 + exp(-data[index]));
			break;
		case ELU:
			if (data[index] < 0)
				data[index] = (exp(data[index]) - 1);
			break;
		case RELIE:
			if (data[index] < 0)
				data[index] *= 0.01;
			break;
		case PLSE:
			if (data[index] < -4)
				data[index] = 0.01 * (data[index] + 4);
			else if (data[index] > 4)
				data[index] = 0.01 * (data[index] - 4) + 1.0;
			else
				data[index] = 0.125 * data[index] + 0.5;
			break;
		case RAMP:
			//return x*(x>0) + 0.1*x;
			val = 0.1 * data[index];
			if (data[index] > 0.0)
				data[index] += val;
			else
				data[index] = val;
			break;
		case STAIR:
		{
			int n = floor(data[index]);
			float t = (float)(n >> 1);
			if (0 == n & 0x01)
				data[index] = t; //if (n % 2 == 0) return floor(x / 2.);
			else
				data[index] = t + (data[index] - n);
			break;
		}
		case LINEAR:
		default:
			break;
		}
		index += threads;
	}

}
#define MIN_DELTA_VAL 1.0e-8
// output is delta
__global__ static void gradient_kernel(float* data, float* delta, int elements, int threads, ACTIVATION_TYPE a) {

	float val;
	int index = blockDim.x  * blockIdx.x + threadIdx.x;
	while (index < elements) {
		switch (a) {
		case LEAKY:
			if (data[index] < 0.0) delta[index] *= 0.1;
			break;
		case LOGISTIC:
			val = data[index] * (1.0 - data[index]);
			delta[index] *= val;
			break;
		case RELU:
			if (data[index] <= 0.0) delta[index] = 0.0;
			break;
		case HARDTAN:
			if (data[index] > -1.0 && data[index] < 1.0) delta[index] = 1.0;
			else delta[index] = 0.0;
			break;
		case LHTAN:
			if (data[index] <= 0.0 || data[index] >= 1.0)
				delta[index] *= 0.001;
			break;
		case TANH:
			val = data[index] * data[index];
			delta[index] *= (1.0 - val);
			break;
		case LOGGY:
			val = (data[index] + 1.0) * 0.5;
			delta[index] = 2.0 * (1 - val) * val * delta[index];
			break;
		case ELU:
			if (data[index] < 0.0)
				delta[index] *= (data[index] + 1.0);
			break;
		case RELIE:
			if (data[index] <= 0) delta[index] *= 0.01;
			break;
		case PLSE:
			if (data[index] < 0 || data[index] > 1)  delta[index] *= 0.01;
			else delta[index] *= 0.125;
			break;
		case RAMP:
			if (data[index] > 0) delta[index] *= 1.1;
			else
				delta[index] *= 0.1;
			break;
		case STAIR:
			if (floor(data[index]) == data[index]) delta[index] = 0.0;
		case LINEAR:
		default:
			break;
		}		
		if (delta[index] < MIN_DELTA_VAL && delta[index] > -MIN_DELTA_VAL)
			delta[index] = 0.0f;
		index += threads;
	}
}
static int dbg_index = 1;
bool gradient_array_ongpu(float *x, float * delta, int n, ACTIVATION_TYPE a) {
 
	if (a == LINEAR) return true;
	int g = GPUGridSize(9999);
	int b = GPUBlockSize(9999);
	int threads = g * b;
	if (n < threads) {
		b = (n + g - 1) / g;
		threads = g * b;
	}
#if 0
	char buffer[200];
	char *cpu_data = NULL;
	size_t bytes;
	ofstream f;
	if (a == LEAKY) { 
		bytes = n * sizeof(float);
		cpu_data = new char[bytes];
		hipError_t err = hipMemcpy(cpu_data, delta, bytes, hipMemcpyDeviceToHost);
		sprintf(buffer, "E:\\AI\\Data\\debugging\\RQNet\\grdient.%02d.before.bin", dbg_index );
		f.open(buffer, ios::trunc); 
		f.write(cpu_data, bytes);
		f.close();
	}
#endif
	gradient_kernel <<<g, b >>>(x, delta , n, threads, a);
	hipError_t err = hipDeviceSynchronize();
#if 0
	if (a == LEAKY) {
		err = hipMemcpy(cpu_data, delta, bytes, hipMemcpyDeviceToHost);
		sprintf(buffer, "E:\\AI\\Data\\debugging\\RQNet\\grdient.%02d.after.bin", dbg_index++);
		f.open(buffer, ios::trunc);
		f.write(cpu_data, bytes);
		f.close();
		delete[]cpu_data;
	}
#endif
	if (err != hipSuccess) {
		cerr << "activation failed!" << endl;
		return false;
	}
	return true;
}
bool print_debugging = false;
bool activate_array_ongpu(float *x, int n, ACTIVATION_TYPE a) {

	int g = GPUGridSize(9999);
	int b = GPUBlockSize(9999);
	int threads = g * b;
	if (n < threads) {
		b = (n + g - 1) / g;
		threads = g * b;
	}
#if 0
	char buffer[200];
	float *cpu_data = NULL;
	ofstream f;
	if (print_debugging) {
		sprintf(buffer, "activation array for 0x%08x , elments : %d, type: %d\n", (unsigned long)x, n, (int)a);
		cout << buffer;
		cpu_data = new float[n];
		hipError_t err = hipMemcpy(cpu_data, x, n * sizeof(float), hipMemcpyDeviceToHost);

		sprintf(buffer, "activation.%08x.txt", (unsigned long)x);
		f.open(buffer, ios::trunc);

		for (int i = 0; i < n; i++) {
			sprintf(buffer, "%.6f ", cpu_data[i]);
			f << buffer;
		}
		f << endl;
	}
#endif
	activate_kernel <<<g, b>>>(x, n, threads, a);
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		cerr << "activation failed!" << endl;
		return false;
	}
#if 0
	if (print_debugging) {
		err = hipMemcpy(cpu_data, x, n * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) {
			sprintf(buffer, "%.6f ", cpu_data[i]);
			f << buffer;
		}
		f << endl;
		f.close();
		delete[]cpu_data;
	}
#endif
	return true;
}