#include "hip/hip_runtime.h"
/*
Each block cannot have more than 512/1024 threads in total (Compute Capability 1.x or
2.x and later respectively)
The maximum dimensions of each block are limited to [512,512,64]/[1024,1024,64]
(Compute 1.x/2.x or later)
Each block cannot consume more than 8k/16k/32k/64k/32k/64k/32k/64k/32k/64k
registers total (Compute 1.0,1.1/1.2,1.3/2.x-
/3.0/3.2/3.5-5.2/5.3/6-6.1/6.2/7.0)
Each block cannot consume more than 16kb/48kb/96kb of shared memory
(Compute 1.x/2.x-6.2/7.0)

For GTX 106:

Total amount of shared memory per block:       49152 bytes
Total number of registers available per block: 65536
Maximum number of threads per multiprocessor:  2048
(10) Multiprocessors, (128) CUDA Cores/MP:     1280 CUDA Cores

Let's use 1280 at most
*/
#include "stdafx.h"
#include "image.h" 


__device__ const float PIXEL_NORM_FACTOR = 1.0 / 255.0;
__global__  void img_post_read_kernel(const byte* src, float* fill, int h, int w, int c) {
	//int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d of %d blocks, %d of %d threads\n", blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);

	//Now this is one of the 1280 threads(10 x 128),
	// and out mission is to fill the mat `fill`
	int stride = h * w;
	int channel_offset = 0;
	for (int i = 0; i < c; i++, channel_offset += stride) {
		for (int y = blockIdx.x; y < h; y += gridDim.x) {
			int offset = w * y;
			for (int x = threadIdx.x; x < w; x += blockDim.x) {
				int dest_index = channel_offset + x + offset;
				int src_index = i + c * (x + offset);
				fill[dest_index] = src[src_index] * PIXEL_NORM_FACTOR;

			}
		}
	}
}

/*
x < 1 : 1 + (a+3) x *x + (a+2)* x * x * x
x < 2 : -4a+8ax-5ax*x+a x*x*x
*/
__device__ static float cubic_interpolation(float x) {
	if (0.0 == x) return 1.0;
	if (x < 0.0) x = 0.0 - x;
	if (1.0 == x || 2.0 <= x) return 0.0;
	float x2 = x * x;
	float x3 = x2 * x;
	if (x < 1.0)
		return 1.0 - 2.5 * x2 + 1.5f * x3;
	return  2.0f - 4.0 * x + 2.5 * x2 - 0.5 * x3;

}


__global__ static void im_resize_kernel_cubic(const float* orig, int orig_height, int orig_width,
	int channels, float* fill, int height, int width, float sh, float sw) {
	float C[4];
	float A[4];
	float T[4];
	int offsets[4], index;
	for (int c = 0, offset_cf = 0, offset_co = 0; c < channels;
		c++, offset_cf += height * width, offset_co += orig_height * orig_width) {
		for (int y = blockIdx.x; y < height; y += gridDim.x) {
			float f_ys = y * sh;
			int ys = (int)floor(f_ys);
			float u = f_ys - ys;
			ys--;
			for (int m = 0; m < 4; m++, ys++) {
				offsets[m] = (ys >= 0 && ys < orig_height) ? (offset_co + ys * orig_width) : -1;
			}
			C[0] = cubic_interpolation(1 + u);
			C[1] = cubic_interpolation(u);
			C[2] = cubic_interpolation(1 - u);
			C[3] = cubic_interpolation(2 - u);
			int offset_f = offset_cf + y * width;
			for (int x = threadIdx.x; x < width; x += blockDim.x) {
				float f_xs = x * sw;
				int xs = floor(f_xs);
				float v = f_xs - xs;
				xs--;

				A[0] = cubic_interpolation(1 + v);
				A[1] = cubic_interpolation(v);
				A[2] = cubic_interpolation(1 - v);
				A[3] = cubic_interpolation(2 - v);
				for (int n = 0; n < 4; n++, xs++) {
					T[n] = 0.0;
					if (xs >= 0 && xs < orig_width) {
						for (int k = 0; k < 4; k++) {
							if (offsets[k] >= 0) {
								index = offsets[k] + xs;
								T[n] += A[k] * orig[index];
							}
						}
					}
				}
				float temp = T[0] * C[0] + T[1] * C[1] + T[2] * C[2] + T[3] * C[3];
				if (temp < 0.0) temp = 0.0;
				else if (temp > 1.0) temp = 1.0;
				fill[offset_f + x] = temp;
			}
		}
	}
}
__device__ static const float dilated_factor = 0.5;

__global__ static void im_resize_kernel_dilated_conv(const float* orig, int orig_width, int orig_height,
	int channels, float* fill, int width, int height, float sw, float sh,  float cfactor) {

	int step_h = (int)roundf(sh * dilated_factor);
	int step_w = (int)roundf(sw * dilated_factor);
	float corder_factor1 = 0.146447 * (1.0 - cfactor);
	float corder_factor2 = 0.103553 * (1.0 - cfactor);
	for (int c = 0, offset_cf = 0, offset_co = 0; c < channels;
		c++, offset_cf += height * width, offset_co += orig_height * orig_width) {
		for (int y = blockIdx.x; y < height; y += gridDim.x) {

			int src_y = (int)roundf(y * sh);
			if (src_y >= orig_height) src_y = orig_height - 1;
			int src_top = src_y - step_h, src_bottom = src_y + step_h;

			if (src_top < 0) src_top = src_y;
			if (src_bottom >= orig_height) src_bottom = src_y;

			int offset_f = offset_cf + y * width,
				offset_s0 = offset_co + src_top * orig_width,
				offset_s1 = offset_co + src_y * orig_width,
				offset_s2 = offset_co + src_bottom * orig_width;

			for (int x = threadIdx.x; x < width; x += blockDim.x) {
				int src_x = (int)roundf(x * sw);
				if (src_x >= orig_width) src_x = orig_width - 1;
				int src_left = src_x - step_w;
				if (src_left < 0) src_left = src_x;
				int src_right = src_x + step_w;
				if (src_right >= orig_width) src_right = src_x;
				fill[offset_f + x] = corder_factor2 * (orig[offset_s0 + src_left] + orig[offset_s0 + src_right] +
					orig[offset_s2 + src_left] + orig[offset_s2 + src_right])
					+ corder_factor1 * (orig[offset_s1 + src_left] + orig[offset_s1 + src_right] +
						orig[offset_s0 + src_x] + orig[offset_s2 + src_x])
					+ cfactor *   orig[offset_s1 + src_x];
			}
		}
	}
}
__global__ static void im_resize_kernel_bilinear(const float* orig, int orig_width, int orig_height,
	int channels, float* fill, int width, int height, float sw, float sh) {
	for (int c = 0, offset_cf = 0, offset_co = 0; c < channels;
		c++, offset_cf += height * width, offset_co += orig_height * orig_width) {
		for (int y = blockIdx.x; y < height; y += gridDim.x) {
			float t = y * sh;
			int src_y0 = (int)floorf(t), src_y1 = src_y0 + 1;
			float v = t - src_y0;
			if (src_y1 >= orig_height) src_y1 = orig_height - 1;
			int offset_f = offset_cf + y * width;
			int offset_s0 = offset_co + src_y0 * orig_width;
			int offset_s1 = offset_co + src_y1 * orig_width;
			for (int x = threadIdx.x; x < width; x += blockDim.x) {
				t = x * sw;
				int src_x0 = (int)floorf(t), src_x1 = src_x0 + 2;
				float u = t - src_x0;
				if (src_x1 >= orig_width) src_x1 = orig_width - 1;
				fill[offset_f + x] = (1.0 - u) * (1.0 - v) * orig[offset_s0 + src_x0]
					+ (1.0 - u) * v * orig[offset_s1 + src_x0]
					+ u * (1.0 - v) *  orig[offset_s0 + src_x1]
					+ u * v *  orig[offset_s1 + src_x1];
			}
		}
	}
}
//Nearest element method
__global__ static void im_resize_kernel_nearest(const float* orig, int orig_width, int orig_height,
	int channels, float* fill, int width, int height, float sw, float sh) {
	for (int c = 0, offset_cf = 0, offset_co = 0; c < channels;
		c++, offset_cf += height * width, offset_co += orig_height * orig_width) {
		for (int y = blockIdx.x; y < height; y += gridDim.x) {
			int src_y = (int)round(y * sh);
			if (src_y >= orig_height) src_y = orig_height - 1;
			int offset_f = offset_cf + y * width;
			int offset_s = offset_co + src_y * orig_width;
			for (int x = threadIdx.x; x < width; x += blockDim.x) {
				int src_x = (int)round(x * sw);
				if (src_x >= orig_width) src_x = orig_width - 1;
				fill[offset_f + x] = orig[offset_s + src_x];
			}
		}
	}
}

bool Image::ResizeTo(int w, int h, bool fast, float center_ratio) {
 
	if (0 == channels || 0 == height || 0 == width ) return false;
	if (w <= 0 || h <= 0) return false;

	if (!PushToGPU()) return false;

	float sh = (float)(height) / (float)h;
	float sw = (float)(width) / (float)w;
	 
	int g = GPUGridSize(h);
	int b = GPUBlockSize(w);


	float* gpu  = NULL;
	int e = w * h * channels;
	
	hipMalloc(&gpu,e * sizeof(float));
	if (!gpu) return false;

	

	if (fast)
		im_resize_kernel_nearest<<<g, b>>>(gpu_data, width, height,channels, gpu, w, h, sw, sh);
	else  if (sh > 2.0 && sw > 2.0) {
		im_resize_kernel_dilated_conv<<<g, b>>>(gpu_data, width, height, channels, gpu, w, h, sw, sh, center_ratio);
	}
	else if (sh < 0.5 && sw < 0.5) {
		im_resize_kernel_cubic<<<g, b>>>(gpu_data, width, height, channels, gpu, w, h, sw, sh);
	}
	else {
		im_resize_kernel_bilinear<<<g, b>>>(gpu_data, width, height, channels, gpu, w, h, sw, sh);
	}

	hipError_t err = hipDeviceSynchronize();

	hipFree(gpu_data);
	if (err != hipSuccess) {
		gpu_data = NULL;
		return false;
	}
	gpu_data = gpu;
	width = w;
	height = h;
	delete[]data;
	data = New float[e];
	return PullFromGPU();
}

__device__ inline float three_way_max(float a, float b, float c) {
	return (a > b) ? ((a > c) ? a : c) : ((b > c) ? b : c);
}

__device__ inline float three_way_min(float a, float b, float c) {
	return (a < b) ? ((a < c) ? a : c) : ((b < c) ? b : c);
}
/*

1: max=max(R,G,B)
2: min=min(R,G,B)
3: if R = max, H = (G-B)/(max-min)
4: if G = max, H = 2 + (B-R)/(max-min)
5: if B = max, H = 4 + (R-G)/(max-min)
6:
7: H = H * 60
8: if H < 0, H = H + 360
9:
10: V=max(R,G,B)
11: S=(max-min)/max
*/
__global__ static void img_rgb2hsv_kernel(float* data, int width, int height,
	float hue, float sat, float val) {
	float r, g, b;
	float h, s, v;


	int offset, r_index, g_index, b_index;
	int channel_elements = height * width;
	for (int y = blockIdx.x; y < height; y += gridDim.x) {
		offset = width * y;
		for (int x = threadIdx.x; x < width; x += blockDim.x) {
			r_index = offset + x;
			g_index = r_index + channel_elements;
			b_index = g_index + channel_elements;
			r = data[r_index];
			g = data[g_index];
			b = data[b_index];
			float max_ = three_way_max(r, g, b);
			float min_ = three_way_min(r, g, b);
			float delta = max_ - min_; // what if delta == 0;
			v = max_; // v=max
			if (0.0f == delta)
				h = 0.0f;
			else if (max_ == r) {
				h = (g - b) / delta;
				if (h < 0) h += 6.0;

			}
			else if (max_ == g) {
				h = 2 + (b - r) / delta;
			}
			else {
				h = 4 + (r - g) / delta;
			}

			if (0.0 == max_)
				s = 0.0;
			else
				s = 1 - min_ / max_;

			h += hue;

			if (h > 6.0f) h -= 6.0f;
			else if (h < 0.0f) h += 6.0f;

			data[r_index] = h;
			data[g_index] = s * sat;
			data[b_index] = v * val;
		}
	}
}
//TODO:
// normalized should be true
bool Image::RGB2HSV(float hue, float sat, float val) {
	if (channels != 3) return false;
	if (!PushToGPU()) return false;
	hue *= 6.0f;
	int g = GPUGridSize(height);
	int b = GPUBlockSize(width);
	img_rgb2hsv_kernel <<<g, b>>>(gpu_data, width, height, hue, sat, val);
	if(hipSuccess != hipDeviceSynchronize())
		return false;
	return PullFromGPU();

}
__device__ static inline float constrain(float x) {
	if (x < 0.0) return 0.0;
	if (x > 1.0) return 1.0;
	return x;
}
__global__ static void img_hsv2rgb_kernel(float* data, int width, int height) {
	float r, g, b;
	float h, s, v;
	float f, p, q, t;
	int offset, r_index, g_index, b_index;
	int channel_elements = height * width;
	for (int y = blockIdx.x; y < height; y += gridDim.x) {
		offset = width * y;
		for (int x = threadIdx.x; x < width; x += blockDim.x) {
			r_index = offset + x;
			g_index = r_index + channel_elements;
			b_index = g_index + channel_elements;
			h = data[r_index];
			s = data[g_index];
			v = data[b_index];
			int index = floor(h);
			f = h - index;
			p = v * (1 - s);
			q = v *(1 - s * f);
			t = v * (1 - s * (1 - f));
			switch (index) {
			case 0:
				r = v; g = t; b = p;
				break;
			case 1:
				r = q; g = v; b = p;
				break;
			case 2:
				r = p; g = v; b = t;
				break;
			case 3:
				r = p; g = q; b = v;
				break;
			case 4:
				r = t; g = p; b = v;
				break;
			case 5:
				r = v; g = p; b = q;
				break;
			default:
				break;
			}

			data[r_index] = constrain(r);
			data[g_index] = constrain(g);
			data[b_index] = constrain(b);
		}
	}
}
bool Image::HSV2RGB() {

	if (channels != 3) return false;
	if (!PushToGPU()) return false; 
	int g = GPUGridSize(height);
	int b = GPUBlockSize(width);
	img_hsv2rgb_kernel <<<g, b >>>(gpu_data, width, height);
	if (hipSuccess != hipDeviceSynchronize())
		return false;
	return PullFromGPU();


}
__global__ static void img_scale_kernel(float* data, int height, int width, float scale) {
	int offset, pixel_idx;
	//int channel_elements = height * width;
	for (int y = blockIdx.x; y < height; y += gridDim.x) {
		offset = width * y;
		for (int x = threadIdx.x; x < width; x += blockDim.x) {
			pixel_idx = offset + x;
			data[pixel_idx] *= scale;
			if (data[pixel_idx] > 1.0) data[pixel_idx] -= 1.0;
			else if (data[pixel_idx] < 0.0) data[pixel_idx] += 1.0;
		}
	}
}
bool Image::Distort(float hue, float sat, float val) {
 
	if (!PushToGPU()) return false;
	int g = GPUGridSize(height);
	int b = GPUBlockSize(width);

	if (channels == 3) {
		img_rgb2hsv_kernel <<<g, b >>>(gpu_data, width, height, hue, sat, val);
		if (hipSuccess != hipDeviceSynchronize())
			return false;
		img_hsv2rgb_kernel <<<g, b >>> (gpu_data, width, height);
		if (hipSuccess != hipDeviceSynchronize())
			return false;
	}
	else {
		img_scale_kernel <<<g, b >>> (gpu_data, width, height,val);
		if (hipSuccess != hipDeviceSynchronize())
			return false;
	}
	return PullFromGPU(); 


}
__global__ static void img_rotate_kernel(float* data, int width, int height, int channels,  RotateType rt, float* transition) {

	int src_index, dest_index;
	int offset, offset1;
	int max_width = width;
	int max_height = height;
	switch (rt) {
	case HorizFlip:
		max_width >>= 1;
		break;
	case VertiFlip:
		max_height >>= 1;
		break;
	default:
		break;

	}
	float temp;
	int c_size = height * width;
	for (int c = 0; c < channels; c++) {
		for (int y = blockIdx.x; y < max_height; y += gridDim.x) {
			offset = y * width;
			if (VertiFlip == rt) offset1 = (height - 1 - y) * width;
			for (int x = threadIdx.x; x < max_width; x += blockDim.x) {
				src_index = offset + x;
				if (HorizFlip == rt) {
					dest_index = offset + (width - 1 - x);
					temp = data[dest_index];
					data[dest_index] = data[src_index];
					data[src_index] = temp;
				}
				else if (VertiFlip == rt) {
					dest_index = offset1 + x;
					temp = data[dest_index];
					data[dest_index] = data[src_index];
					data[src_index] = temp;
				}
				else if (ToLeft == rt) {
					dest_index = (width - 1 - x) * height + y;
					transition[dest_index] = data[src_index];

				}
				else if (ToRight == rt) { //
					dest_index = x * height + (height - 1 - y);
					transition[dest_index] = data[src_index];
				}
				else { // Rotate 180
					;
				}
			}
		}
		data += c_size;
		if(ToLeft == rt || ToRight == rt)
			transition += c_size;
	}
}
__global__ static void img_rot180_kernel(float* data, int width, int height, int channels, int elements) {

	int max_y = height >> 1; 
	int c_size = height * width;
	
	int y = blockIdx.x;
	int x = threadIdx.x; 
	for (int c = 0 ; c < channels; c++, data += c_size) {
		while (y < max_y) {
			int ny = height - y - 1;
			int off = y * width;
			int n_off = ny * width;
			while (x < width) {
				int nx = width - x - 1;
				int idx = off + x;
				int n_idx = n_off + nx;
				if (idx < c_size && n_idx < c_size) {
					int temp = data[idx];
					data[idx] = data[n_idx];
					data[n_idx] = temp;
				} 
				x += blockDim.x;
			}
			y += gridDim.x;
		}
	}
}
bool Image::Rotate(RotateType rt) {

	if (!PushToGPU()) return false;
	int g = GPUGridSize(height);
	int b = GPUBlockSize(width);

	float* transition = NULL;  
	int e = height * width * channels;
	hipError_t err;
	if (ToLeft == rt || ToRight == rt) {
		err = hipMalloc(&transition, e * sizeof(float));
		if (hipSuccess != err) return false;
		int temp = height;
		height = width;
		width = temp;
	}
	if (rt == Rotate180)
		img_rot180_kernel<<<g,b>>>(gpu_data, width, height, channels, e);
	else
		img_rotate_kernel<<<g,b>>>(gpu_data,  width, height, channels, rt, transition);

	err = hipDeviceSynchronize();

	if (ToLeft == rt || ToRight == rt) {
		hipMemcpy(gpu_data, transition, e * sizeof(float), hipMemcpyDeviceToDevice);
		hipFree(transition);
	}
	
	if (err != hipSuccess)
		return false;
	return PullFromGPU(); 
}

